#include <iostream>

#include <hip/hip_runtime.h>
#include ""
#include "hip/hip_runtime.h"

#include <pqxx/pqxx>
#include <string>
#include <vector>


using namespace pqxx;
using namespace std;


result giveResult(connection &C,  string sql)
{
	nontransaction N(C);
	result R(N.exec( sql ));
	return R;
}


int main()
{

	string sql;
	vector<string> Schema;	
	
	int v1;
	int v2;

	cout << "Hello World!" << endl;
	try{
		connection C("dbname=name user=user password=password hostaddr=ipaddress port=5432");
		if (C.is_open())
		{
			cout << "Database " << C.dbname()  << " opened" << endl;
			
		}
		else
		{
			cout << "Cannot Open Database " << endl;
			return 1;
		}

		sql = "SELECT * "\
		      " FROM firing_times_test0 LIMIT 1;";

		//nontransaction N(C);

		//result R( N.exec( sql ));
		
		result R = giveResult(C, sql);

		/*
		for (result::const_iterator c = R.begin(); c != R.end(); ++c)
		{
			std::cout << (*c).size() << std::endl;
			c[0].to(v1);
			c[1].to(v2);
			cout << typeid(v1).name() << ":"  << v1  << "," << typeid(v2).name() << ":" <<  v2 <<  endl;
						
		}
		
		*/

		result::const_iterator c = R.begin();

		cout << c[0] << "," << c[1] << endl;

		C.disconnect();
	}
	catch(const exception &e)
	{
		cerr << e.what() << endl;
		return 1;
	}
	


	return 0;
}





