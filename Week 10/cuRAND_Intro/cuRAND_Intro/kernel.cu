
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <ctime>
#include <iostream>
#include <iomanip>

#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"

__global__ void initializePRG(
	int size,
	unsigned int seed,
	hiprandState_t *d_states)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= size){return;}
	hiprand_init(
		seed,
		i,
		0,
		&d_states[i]);
}


__global__ void uniformDist(
	int size,
	hiprandState_t *d_states,
	float *d_values)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= size){ return; }
	d_values[i] = hiprand_uniform(&d_states[i]);
}


__global__ void normalDist(
	int size,
	hiprandState_t *d_states,
	float *d_values)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= size){ return; }
	d_values[i] = hiprand_normal(&d_states[i]);
}


int main()
{
	unsigned int N = 100000;
	unsigned int numThreads = 256;

	unsigned int numBlocks = (N + numThreads - 1) / numThreads;

	hiprandState_t *d_states;
	hipMalloc((void**)&d_states, N * sizeof(hiprandState_t));

	initializePRG <<< numBlocks , numThreads >>>(
		N, time(NULL), d_states);

	float *h_values = new float[N];
	for (int i = 0; i < N; i++)
	{
		h_values[i] = 0.0f;
	}

	float *d_values;

	hipMalloc((void**)&d_values, N * sizeof(float));

	uniformDist <<< numBlocks, numThreads >>>(
		N, d_states, d_values);

	hipMemcpy(h_values, d_values, N * sizeof(float), hipMemcpyDeviceToHost);

	std::cout << std::setprecision(4);
	for (int i = 0; i < 100; i++)
	{
		std::cout << h_values[i] << " ";
		if ((i + 1) % 50 == 0)
		{
			std::cout << std::endl;
		}
	}

	normalDist <<< numBlocks, numThreads >>>(
		N, d_states, d_values);

	hipMemcpy(h_values, d_values, N * sizeof(float), hipMemcpyDeviceToHost);


	hipFree(d_values);
	hipFree(d_states);
	delete[] h_values;

    return 0;
}
