
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <ctime>
#include <iostream>

#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"


__global__ void initializePRG(
	int size, unsigned int seed, hiprandState_t *d_states)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= size){ return; }
	hiprand_init(
		seed,
		i,
		0,
		&d_states[i]);
}


__global__ void monteCarlo(
	int size, hiprandState_t *d_states, float *d_v,
	float mu, float sigma,
	int T, float dt)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= size){ return; }
	for (int t = 0; t < T; t++)
	{
		d_v[i] += d_v[i]*(dt*mu + sqrtf(dt)*hiprand_normal(&d_states[i])*sigma);
	}
}


int main()
{
	unsigned int N = 100000;
	unsigned int numThreads = 256;
	unsigned int numBlocks = (N + numThreads - 1) / numThreads;
	float mu = 0.1;
	float sigma = 0.2;

	int T = 252;
	float dt = 1 / 252.0f;

	float *h_traj = new float[N];
	for (int i = 0; i < N; i++)
	{
		h_traj[i] = 100.0f;
	}

	float *d_traj;
	hipMalloc((void**)&d_traj, N*sizeof(float));

	hipMemcpy(d_traj, h_traj, N * sizeof(float), hipMemcpyHostToDevice);

	hiprandState_t *d_states;
	hipMalloc((void**)&d_states, N * sizeof(hiprandState_t));

	initializePRG <<< numBlocks, numThreads >>>(
		N, time(NULL), d_states);

	monteCarlo <<< numBlocks, numThreads >>>(
		N, d_states, d_traj,
		mu, sigma,
		T, dt);

	hipMemcpy(h_traj, d_traj, N * sizeof(float), hipMemcpyDeviceToHost);

	float avgEnd = 0.0f;

	for (int i = 0; i < N; i++)
	{
		//std::cout << h_traj[i] << std::endl;
		avgEnd += h_traj[i];
	}
	avgEnd /= float(N);
	std::cout << "The Average Value Is " << avgEnd << std::endl;

	delete[] h_traj;
	hipFree(d_states); hipFree(d_traj);

    return 0;
}
